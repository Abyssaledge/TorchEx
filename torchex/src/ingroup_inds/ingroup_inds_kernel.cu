#include "hip/hip_runtime.h"
#include <assert.h>
#include <vector>
#include <math.h>
#include <stdio.h>
#include <torch/serialize/tensor.h>
#include <torch/extension.h>
#include <torch/types.h>
#include "hip/hip_fp16.h"
#include "../utils/error.cuh"

#define THREADS_PER_BLOCK 256
#define DIVUP(m, n) ((m) / (n) + ((m) % (n) > 0))

// #define DEBUG
// #define ASSERTION

__global__ void ingroup_inds_kernel(
    const long *group_inds,
    long *out_inds,
    int *ingroup_counter,
    int N
) {

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= N) return;
  long this_group_id = group_inds[idx];

  int cnt = atomicAdd(&ingroup_counter[this_group_id], 1);
  out_inds[idx] = cnt;
}


 void ingroup_inds_launcher(
  const long *group_inds,
  long *out_inds,
  int N,
  int max_group_id
  ) {

  int *ingroup_counter = NULL;
  CHECK_CALL(hipMalloc(&ingroup_counter,   (max_group_id + 1) * sizeof(int)));
  CHECK_CALL(hipMemset(ingroup_counter, 0, (max_group_id + 1) * sizeof(int)));

  dim3 blocks(DIVUP(N, THREADS_PER_BLOCK));
  dim3 threads(THREADS_PER_BLOCK);

  ingroup_inds_kernel<<<blocks, threads>>>(
      group_inds,
      out_inds,
      ingroup_counter,
      N
  );

  hipFree(ingroup_counter);

  #ifdef DEBUG
  CHECK_CALL(hipGetLastError());
  CHECK_CALL(hipDeviceSynchronize());
  #endif

  return;

}
