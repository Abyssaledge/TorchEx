#include "hip/hip_runtime.h"
#include "../utils/error.cuh"
#include "../utils/timer.cuh"
#include <assert.h>
#include <cfloat>
#include <fstream>
#include <sstream>
#include <vector>

#define MAX_THREADS 1024
#define THREADS_PER_BLOCK 256
#define WARP_SIZE 32
#define DIVUP(m, n) ((m + n - 1) / n)

__forceinline__ int up_2n(int n) {
    if (n == 1)
        return 1;
    int temp = n - 1;
    temp |= temp >> 1;
    temp |= temp >> 2;
    temp |= temp >> 4;
    temp |= temp >> 8;
    temp |= temp >> 16;
    return temp + 1;
}

template <typename T>
__device__ inline T warpReduceSum(T sum, int blockSize) {
    if (blockSize >= 32)
        sum += __shfl_down_sync(0xffffffff, sum, 16);
    if (blockSize >= 16)
        sum += __shfl_down_sync(0xffffffff, sum, 8);
    if (blockSize >= 8)
        sum += __shfl_down_sync(0xffffffff, sum, 4);
    if (blockSize >= 4)
        sum += __shfl_down_sync(0xffffffff, sum, 2);
    if (blockSize >= 2)
        sum += __shfl_down_sync(0xffffffff, sum, 1);
    return sum;
}

template <typename T>
__device__ inline void warpReduceMax(T &max_value, int &idx, int blockSize) {
    if (blockSize >= 32) {
        T temp_max = __shfl_down_sync(0xffffffff, max_value, 16);
        int temp_idx = __shfl_down_sync(0xffffffff, idx, 16);
        if (temp_max > max_value) {
            max_value = temp_max;
            idx = temp_idx;
        }
    }
    if (blockSize >= 16) {
        T temp_max = __shfl_down_sync(0xffffffff, max_value, 8);
        int temp_idx = __shfl_down_sync(0xffffffff, idx, 8);
        if (temp_max > max_value) {
            max_value = temp_max;
            idx = temp_idx;
        }
    }
    if (blockSize >= 8) {
        T temp_max = __shfl_down_sync(0xffffffff, max_value, 4);
        int temp_idx = __shfl_down_sync(0xffffffff, idx, 4);
        if (temp_max > max_value) {
            max_value = temp_max;
            idx = temp_idx;
        }
    }
    if (blockSize >= 4) {
        T temp_max = __shfl_down_sync(0xffffffff, max_value, 2);
        int temp_idx = __shfl_down_sync(0xffffffff, idx, 2);
        if (temp_max > max_value) {
            max_value = temp_max;
            idx = temp_idx;
        }
    }
    if (blockSize >= 2) {
        T temp_max = __shfl_down_sync(0xffffffff, max_value, 1);
        int temp_idx = __shfl_down_sync(0xffffffff, idx, 1);
        if (temp_max > max_value) {
            max_value = temp_max;
            idx = temp_idx;
        }
    }
}

__global__ void getPreSum(const int *const unq_inv, int *const preSum, int n) {
    extern __shared__ int groupIdx[];
    int tid = threadIdx.x;
    int i = tid + blockIdx.x * blockDim.x;
    groupIdx[tid] = (i < n) ? unq_inv[i] : -1;
    __syncthreads();
    int groupIdx_i = -1, groupIdx_i_ = -1;
    if (i < n - 1) {
        groupIdx_i = groupIdx[tid];
        groupIdx_i_ = (tid == blockDim.x - 1) ? unq_inv[i + 1] : groupIdx[tid + 1];
    } else if (i == n - 1) {
        groupIdx_i_ = groupIdx_i + 1; // make them unequal
    }
    if (groupIdx_i != groupIdx_i_)
        preSum[groupIdx[tid] + 1] = i + 1;
}

__global__ void scatter_sum(const float *const d_feats, const int *const d_preSum, float *const d_out, int num_unq, int num_dim) {
    int unq_idx = threadIdx.y + blockIdx.y * blockDim.y;
    int tid = threadIdx.x;
    int dim = blockIdx.x;
    extern __shared__ float warpMax[];
    int num_valid_warp = DIVUP(blockDim.x, WARP_SIZE);
    float sum = 0;
    int begin = -1, end = -1;
    assert(dim < num_dim);
    if (unq_idx < num_unq) {
        begin = d_preSum[unq_idx], end = d_preSum[unq_idx + 1];
    }
    for (int feat_idx = begin + tid; feat_idx < end; feat_idx += blockDim.x) {
        sum += d_feats[feat_idx * num_dim + dim];
    }
    int laneIdx = tid % WARP_SIZE;
    int warpIdx = tid / WARP_SIZE;
    sum = warpReduceSum(sum, blockDim.x);
    if (laneIdx == 0)
        warpMax[threadIdx.y * num_valid_warp + warpIdx] = sum;
    __syncthreads();
    sum = (tid < num_valid_warp) ? warpMax[threadIdx.y * num_valid_warp + tid] : 0;
    if (warpIdx == 0)
        sum = warpReduceSum(sum, num_valid_warp);
    if (tid == 0 && unq_idx < num_unq) {
        d_out[unq_idx * num_dim + dim] = sum;
    }
}

__global__ void scatter_max(const float *const d_feats, const int *const d_preSum, float *const d_out, int *const d_arg, int num_unq, int num_dim) {
    int unq_idx = threadIdx.y + blockIdx.y * blockDim.y;
    int tid = threadIdx.x;
    int dim = blockIdx.x;
    int num_valid_warp = DIVUP(blockDim.x, WARP_SIZE);
    extern __shared__ float shared_mem[];
    float *warpMax = shared_mem;
    int *warpMaxIdx = (int *)&warpMax[blockDim.y * num_valid_warp];
    float max_value = -FLT_MAX;
    int max_idx = -1;
    int begin = -1, end = -1;
    assert(dim < num_dim);
    if (unq_idx < num_unq) {
        begin = d_preSum[unq_idx], end = d_preSum[unq_idx + 1];
    }
    for (int feat_idx = begin + tid; feat_idx < end; feat_idx += blockDim.x) {
        float temp_feat = d_feats[feat_idx * num_dim + dim];
        if (temp_feat >= max_value) {
            max_value = temp_feat;
            max_idx = feat_idx;
        }
    }
    int laneIdx = tid % WARP_SIZE;
    int warpIdx = tid / WARP_SIZE;
    warpReduceMax(max_value, max_idx, blockDim.x);
    if (laneIdx == 0) {
        warpMax[threadIdx.y * num_valid_warp + warpIdx] = max_value;
        warpMaxIdx[threadIdx.y * num_valid_warp + warpIdx] = max_idx;
    }
    __syncthreads();
    if (tid < num_valid_warp) {
        max_value = warpMax[threadIdx.y * num_valid_warp + tid];
        max_idx = warpMaxIdx[threadIdx.y * num_valid_warp + tid];
    }
    if (warpIdx == 0)
        warpReduceMax(max_value, max_idx, num_valid_warp);
    if (tid == 0 && unq_idx < num_unq) {
        d_out[unq_idx * num_dim + dim] = max_value;
        d_arg[unq_idx * num_dim + dim] = max_idx;
    }
}

void getPreSum_launcher(const int *const unq_inv, int *const preSum, int num_total) {
    getPreSum<<<DIVUP(num_total, THREADS_PER_BLOCK), THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(int)>>>(unq_inv, preSum, num_total);
}

void scatter_sum_launcher(const float *const feats, const int *const preSum, float *const out,
                          int channel, int num_unq, int max_cnt) {
    int max_2n = max(min(up_2n(max_cnt), MAX_THREADS), 32);
    dim3 blockSize(max_2n, MAX_THREADS / max_2n);
    dim3 gridSize(channel, DIVUP(num_unq, blockSize.y));
    scatter_sum<<<gridSize, blockSize, blockSize.y * DIVUP(max_2n, WARP_SIZE) * sizeof(float)>>>(feats, preSum, out, num_unq, channel);
}

void scatter_max_launcher(const float *const feats, const int *const preSum, float *const out, int *const arg,
                          int channel, int num_unq, int max_cnt) {
    int max_2n = max(min(up_2n(max_cnt), MAX_THREADS), 32);
    dim3 blockSize(max_2n, MAX_THREADS / max_2n);
    dim3 gridSize(channel, DIVUP(num_unq, blockSize.y));
    int shared_mem = blockSize.y * DIVUP(max_2n, WARP_SIZE) * (sizeof(float) + sizeof(int));
    scatter_max<<<gridSize, blockSize, shared_mem>>>(feats, preSum, out, arg, num_unq, channel);
}

// void read_file(std::string filename, std::vector<int> &array, int num_cols) {
//     std::ifstream infile(filename.c_str());
//     std::string line;
//     int word;
//     if (!infile) {
//         printf("Cannot open test_data.txt");
//         exit(1);
//     }
//     while (std::getline(infile, line)) {
//         std::istringstream words(line);
//         if (line.length() == 0) {
//             continue;
//         }
//         for (int i = 0; i < num_cols; i++) {
//             if (words >> word) {
//                 array.push_back(word);
//             } else {
//                 printf("Error for reading test_data.txt\n");
//                 exit(1);
//             }
//         }
//     }
//     infile.close();
// }

// int main() {
//     // input
//     std::vector<int> unq_preSum, unq_inv;
//     std::string preSum_name = "/home/yangyuxue/CudaPractice/connect/unq_preSum_test.txt", inv_name = "/home/yangyuxue/CudaPractice/connect/unq_inv_test.txt";
//     int num_cols = 1;
//     read_file(preSum_name, unq_preSum, num_cols);
//     read_file(inv_name, unq_inv, num_cols);
//     int channel = 128;
//     int num_unq = unq_preSum.size() - 1;
//     int num_total = unq_inv.size();
//     // int *calc_preSum = new int[num_unq + 1];
//     int *d_unq_inv;
//     // int *d_preSum, *d_max_cnt;
//     // int *max_cnt = new int(0);
//     int inv_mem = num_total * num_cols * sizeof(int);
//     // int preSum_mem = (num_unq + 1) * num_cols * sizeof(int);
//     CHECK_CALL(hipMalloc(&d_unq_inv, inv_mem));
//     CHECK_CALL(hipMemcpy(d_unq_inv, unq_inv.data(), inv_mem, hipMemcpyHostToDevice));
//     // CHECK_CALL(hipMalloc(&d_preSum, preSum_mem));
//     // CHECK_CALL(hipMalloc(&d_max_cnt, sizeof(int)));
//     // CHECK_CALL(hipMemset(d_max_cnt, 0, sizeof(int)));
//     float *d_feats, *feats = new float[num_total * channel];
//     for (int i = 0; i < num_total * channel; i++)
//         feats[i] = 1.0;
//     uint feats_mem = num_total * channel * sizeof(float);
//     CHECK_CALL(hipMalloc(&d_feats, feats_mem));
//     CHECK_CALL(hipMemcpy(d_feats, feats, feats_mem, hipMemcpyHostToDevice));
//     float *d_out, *out = new float[num_unq * channel];
//     CHECK_CALL(hipMalloc(&d_out, num_unq * channel * sizeof(float)));

//     GPUTimer timer;
//     timer.start();
//     scatter_sum_launcher(d_feats, d_unq_inv, d_out, num_total, channel, num_unq);
//     timer.stop();
//     CHECK_CALL(hipMemcpy(out, d_out, num_unq * channel * sizeof(float), hipMemcpyDeviceToHost));
//     for (int i = 0; i < num_unq; i++) {
//         float delta = unq_preSum[i + 1] - unq_preSum[i];
//         for (int j = 0; j < channel; j++) {
//             if (abs(out[i * channel] - delta) > 1e-3)
//                 printf("error. out[%3d][%3d]:%3.1f, cnt[i]:%3.1f\n", i, j, out[i * channel + j], delta);
//         }
//     }
//     return 0;
// }